#include "hip/hip_runtime.h"
#include "stdio.h"

#include <LexRisLogic/Convert.h>

#include "GPURAM.h"

#include "GPU/RadixSort.h"
#include "Scene.h"
#include "GPU/Interval.h"
#include "GPU/IntervalTree.h"

/***GPU SAP***/
__device__ void sort(float* input,unsigned int output[], int begin, int size,bool x_y)
{
    int axes=(x_y)?3:1;
    int last=0;
    for(int i=0;i<size;++i)
    {
        for(int a=0;a<2;++a)
        {
            int asignacion=0;
            int real_id=(begin+i)*5+a+axes;
            for(int j=0;j<last;++j)
            {
                int real=output[j];
                if(input[real]>input[real_id])
                {
                    break;
                }
                ++asignacion;
            }
            for(int d=last;d>asignacion;--d)
            {
                output[d]=output[d-1];
            }
            output[asignacion]=real_id;
            ++last;
        }
    }
}

__device__ unsigned int get_id(unsigned int i,unsigned int j,unsigned int size)
{
    unsigned int new_index=0;
    if(i>j)
    {
        for(unsigned int k=size-1;k>i;--k)
            new_index+=k;
        return new_index+j;
    }
    else
    {
        for(unsigned int k=size-1;k>j;--k)
            new_index+=k;
        return new_index+i;
    }
}

__global__ void cuda_sap_gpu_parallel( float* objects,int* sizes,
									   char* output,//__local char* sub_coll,
									   const unsigned int parallel_x, const unsigned int parallel_y,
									   const unsigned int total_per_thread,unsigned int max_data_elemets,
									   const unsigned int local_size)
{
	int xindex = threadIdx.x + blockIdx.x * blockDim.x;
	int yindex = threadIdx.y + blockIdx.y * blockDim.y;
	int index = xindex + (gridDim.x * gridDim.y * yindex);
    int max_data=parallel_x*parallel_y;
    int begin_index = index*total_per_thread;
    int end_index = (index+1)*total_per_thread;
    if(begin_index >= max_data)
        return;
    int size_begin_real=0;
    for(int i=0;i<begin_index && i<max_data;++i)
    {
        size_begin_real+=sizes[i];
    }
    unsigned int* ids =(unsigned int*) malloc(local_size*2*sizeof(unsigned int));   //local_size*2
    char* sub_coll = (char*) malloc((local_size*(local_size-1)/2)*sizeof(char));      //local_size*(local_size-1)/2
    char* in_use = (char*) malloc(local_size*sizeof(char));        //local_size
    for(int i=begin_index;i<end_index && i<max_data;++i)
    {
        if(sizes[i]>1)
        {
            int limit=sizes[i]*(sizes[i]-1)/2;
            for(int j=0;j<sizes[i];++j)
            {
                in_use[j]=0;
            }
            for(int j=0;j<limit;++j)
            {
                sub_coll[j]=0;
            }
            sort(objects,ids,size_begin_real,sizes[i],false);
            for(int j=0;j<sizes[i]*2;++j)
            {
                int real_id=ids[j];
                if(real_id%5==1)
                {
                    for(int l=0;l<sizes[i];++l)
                    {
                        if(in_use[l])
                        {
                            int pi=real_id/5-size_begin_real;
                            int pj=l;
                            sub_coll[get_id(pi,pj,sizes[i])]=1;
                        }
                    }
                    in_use[real_id/5-size_begin_real]=1;
                }
                else if(real_id%5==2)
                {
                    in_use[real_id/5-size_begin_real]=0;
                }
            }
            for(int j=0;j<sizes[i];++j)
                in_use[j]=0;
            sort(objects,ids,size_begin_real,sizes[i],true);
            for(int j=0;j<sizes[i]*2;++j)
            {
                int real_id=ids[j];
                if(real_id%5==3)
                {
                    for(int l=0;l<sizes[i];++l)
                    {
                        if(in_use[l])
                        {
                            int pi=real_id/5-size_begin_real;
                            int pj=l;
                            if(sub_coll[get_id(pi,pj,sizes[i])])
                            {
                                int id_a=(pi+size_begin_real)*5;
                                int id_b=(pj+size_begin_real)*5;
                                int i_a=objects[id_a];
                                int i_b=objects[id_b];
                                output[get_id(i_a,i_b,max_data_elemets)]=1;
                            }
                        }
                    }
                    in_use[real_id/5-size_begin_real]=1;
                }
                else if(real_id%5==4)
                {
                    in_use[real_id/5-size_begin_real]=0;
                }
            }
        }
        size_begin_real+=sizes[i];
    }
}
/***GPU SAP***/


/***GPU LL INTERVAL TREE SAP***/
__global__ void cuda_sap_ll_itree_gpu_parallel(float* objects,int* sizes,
											   char* output,//__local char* sub_coll,
											   const unsigned int parallel_x, const unsigned int parallel_y,
											   const unsigned int total_per_thread,unsigned int max_data_elemets,
											   const unsigned int local_size)
{
	int xindex = threadIdx.x + blockIdx.x * blockDim.x;
	int yindex = threadIdx.y + blockIdx.y * blockDim.y;
	int index = xindex + (gridDim.x * gridDim.y * yindex);
    int max_data=parallel_x*parallel_y;
    int begin_index = index*total_per_thread;
    int end_index = (index+1)*total_per_thread;
    if(begin_index >= max_data)
        return;
    int size_begin_real=0;
    for(int i=0;i<begin_index && i<max_data;++i)
    {
        size_begin_real+=sizes[i];
    }
    for(int i=begin_index;i<end_index && i<max_data;++i)
    {
        if(sizes[i]>1)
        {
            int n=sizes[i];
            int n_2=sizes[i]*2;
            float* dataX=CreateArray(objects,sizes[i],false,size_begin_real);
            unsigned short* iX=RadixSort(dataX,n_2);
            free(dataX);
            unsigned short* R=(unsigned short*)malloc(n_2*sizeof(unsigned short));
            for(unsigned short Ri=0;Ri<n_2;++Ri)
                R[iX[Ri]]=Ri;
            float* dataY=CreateArray(objects,sizes[i],true,size_begin_real);
            unsigned short* iY=RadixSort(dataY,n_2);
            ///Algorithm
            IntervalTree S=IntervalTree_create();
            for(int i=0;i<n_2;++i)
            {
                unsigned short p=iY[i];
                if(p<n)
                {
                    struct CInterval intervalo=Interval_create(R[p],R[p+n]);
                    struct List result=IntervalTree_range_query(&S,intervalo);
                    for(struct ListNode* d=result.root;d;d=d->next)
                    {
                        int i_a=dataY[p+n_2];
                        int i_b=dataY[(unsigned int)(iX[(unsigned int)(d->interval.data[0])]+n_2)];
                        output[get_id(i_a,i_b,max_data_elemets)]=1;
                    }
                    List_clear(&result);
                    IntervalTree_insert(&S,intervalo);
                }
                else
                {
                    IntervalTree_remove(&S,Interval_create(R[p-n],R[p]));
                }
            }
            IntervalTree_clear(&S);
            free(iY);
            free(dataY);
            free(iX);
        }
        size_begin_real+=sizes[i];
    }
}
/***GPU LL INTERVAL TREE SAP***/

char* gpu_results=nullptr;
char* gpu_ram_results=nullptr;
float* input_buffer;
int* ref_buffer;
char* output_buffer;
char* ram_buffer;
size_t size_cuda_heap=256;
size_t size_ram=1*1024*1024;

void build_sap_gpu_parallel(int local_size,unsigned int total_objects,unsigned int total_real_objects, unsigned int size_x, unsigned int size_y)
{
    unsigned int max_outputs = (total_objects * (total_objects-1))/2;
    gpu_results=new char[max_outputs];
	hipMalloc((void**)&input_buffer, total_real_objects * 5 * sizeof(float));
	hipMalloc((void**)&ref_buffer, size_x*size_y * sizeof(int));
	hipMalloc((void**)&output_buffer, max_outputs * sizeof(char));
}

void build_sap_ram_gpu_parallel(int local_size,unsigned int total_objects,unsigned int total_real_objects, unsigned int size_x, unsigned int size_y, unsigned int threads)
{
    unsigned int total_per_thread=std::ceil(1.0*size_x*size_y/threads);
    unsigned int max_outputs = (total_objects * (total_objects-1))/2;
    int threads_func=1.0*size_x*size_y/total_per_thread;
    gpu_ram_results=new char[max_outputs];
	hipMalloc((void**)&input_buffer, total_real_objects * 5 * sizeof(float));
	hipMalloc((void**)&ref_buffer, size_x*size_y * sizeof(int));
	hipMalloc((void**)&output_buffer, max_outputs * sizeof(char));
	hipMalloc((void**)&ram_buffer, threads_func * size_ram * sizeof(char));
}

bool SAP_GPU_Parallel(float* objects,
                      int* total_sizes,
                      unsigned int total_objects,
                      std::vector<int>& total_collision,
                      std::list<std::pair<int,int>>& collision,
                      float* time_construction,float* time_collision,
                      int threads,unsigned int size_x,unsigned int size_y,
                      unsigned int total_real_objects,
                      unsigned int local_size)
{
    LL::Chronometer chronometer;
    chronometer.play();
    //Construction BEGIN
    unsigned int max_outputs = (total_objects * (total_objects-1))/2;
    for(unsigned int i=0;i<max_outputs;++i)
        gpu_results[i]=0;
	hipMemcpy( input_buffer, objects, total_real_objects * 5 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( ref_buffer, total_sizes, size_x*size_y * sizeof(int), hipMemcpyHostToDevice);
    unsigned int total_per_thread=std::ceil(1.0*size_x*size_y/threads);
    //Construction END
    chronometer.stop();
    if(time_construction)
        *time_construction=chronometer.get_time();
    chronometer.play();
    //Collision BEGIN
    size_t local=1;
    size_t global=threads;
	cuda_sap_gpu_parallel<<<global,local>>>(input_buffer,ref_buffer,output_buffer,
											size_x,size_y,
											total_per_thread,total_objects,
											local_size);
	hipDeviceSynchronize();
    hipError_t err;
    err = hipGetLastError();
    if (err != hipSuccess)
	{
        printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}
    chronometer.stop();
    if(time_collision)
        *time_collision=chronometer.get_time();
    chronometer.play();
    hipMemcpy( gpu_results, output_buffer, max_outputs*sizeof(char), hipMemcpyDeviceToHost);
    unsigned int index_a=total_objects-1;
    unsigned int index_b=0;
    for(unsigned int i=0;i<max_outputs;++i)
    {
		// std::cout<<"Data "<<i<<": "<<gpu_results[i]<<std::endl;
        if(gpu_results[i])
        {
            total_collision[index_a]+=1;
            total_collision[index_b]+=1;
            collision.push_back(std::pair<int,int>(index_a,index_b));
        }
        ++index_b;
        if(index_a==index_b)
        {
            --index_a;
            index_b=0;
        }
    }
    //Collision END
    chronometer.stop();
    if(time_construction)
        *time_construction+=chronometer.get_time();
	return true;
}

bool SAP_GPU_LL_ITree_Parallel(float* objects,
							   int* total_sizes,
							   unsigned int total_objects,
							   std::vector<int>& total_collision,
							   std::list<std::pair<int,int>>& collision,
							   float* time_construction,float* time_collision,
							   int threads,unsigned int size_x,unsigned int size_y,
							   unsigned int total_real_objects,
							   unsigned int local_size)
{
    LL::Chronometer chronometer;
    chronometer.play();
    //Construction BEGIN
    unsigned int max_outputs = (total_objects * (total_objects-1))/2;
    for(unsigned int i=0;i<max_outputs;++i)
        gpu_results[i]=0;
	hipMemcpy( input_buffer, objects, total_real_objects * 5 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( ref_buffer, total_sizes, size_x*size_y * sizeof(int), hipMemcpyHostToDevice);
    unsigned int total_per_thread=std::ceil(1.0*size_x*size_y/threads);
    //Construction END
    chronometer.stop();
    if(time_construction)
        *time_construction=chronometer.get_time();
    chronometer.play();
    //Collision BEGIN
    size_t local=1;
    size_t global=threads;
	cuda_sap_ll_itree_gpu_parallel<<<global,local>>>(input_buffer,ref_buffer,output_buffer,
													 size_x,size_y,
													 total_per_thread,total_objects,
													 local_size);
	hipDeviceSynchronize();
    hipError_t err;
    err = hipGetLastError();
    if (err != hipSuccess)
	{
        printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}
    chronometer.stop();
    if(time_collision)
        *time_collision=chronometer.get_time();
    chronometer.play();
    hipMemcpy( gpu_results, output_buffer, max_outputs*sizeof(char), hipMemcpyDeviceToHost);
    unsigned int index_a=total_objects-1;
    unsigned int index_b=0;
    for(unsigned int i=0;i<max_outputs;++i)
    {
		// std::cout<<"Data "<<i<<": "<<int(gpu_results[i])<<std::endl;
        if(gpu_results[i])
        {
            total_collision[index_a]+=1;
            total_collision[index_b]+=1;
            collision.push_back(std::pair<int,int>(index_a,index_b));
        }
        ++index_b;
        if(index_a==index_b)
        {
            --index_a;
            index_b=0;
        }
    }
    //Collision END
    chronometer.stop();
    if(time_construction)
        *time_construction+=chronometer.get_time();
	return true;
}


unsigned int RAM_non_getTotalBytes(unsigned int ramBytes)
{
    return ramBytes+std::ceil(ramBytes/8.0);
}

bool SAP_GPU__RAM_LL_ITree_Parallel(float* objects,
									int* total_sizes,
									unsigned int total_objects,
									std::vector<int>& total_collision,
									std::list<std::pair<int,int>>& collision,
									float* time_construction,float* time_collision,
									int threads,unsigned int size_x,unsigned int size_y,
									unsigned int total_real_objects,
									unsigned int local_size)
{
    LL::Chronometer chronometer;
    chronometer.play();
    //Construction BEGIN
    unsigned int max_outputs = (total_objects * (total_objects-1))/2;
    for(unsigned int i=0;i<max_outputs;++i)
        gpu_ram_results[i]=0;
	hipMemcpy( input_buffer, objects, total_real_objects * 5 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( ref_buffer, total_sizes, size_x*size_y * sizeof(int), hipMemcpyHostToDevice);
    unsigned int total_per_thread=std::ceil(1.0*size_x*size_y/threads);
    //Construction END
    chronometer.stop();
    if(time_construction)
        *time_construction=chronometer.get_time();
    chronometer.play();
    //Collision BEGIN
    size_t local=1;
    size_t global=threads;
	cuda_sap_ram_ll_itree_gpu_parallel<<<global,local>>>(input_buffer,ref_buffer,output_buffer,ram_buffer,
														 size_x,size_y,
														 total_per_thread,total_objects,
														 RAM_non_getTotalBytes(size_ram),size_ram);
	hipDeviceSynchronize();
    hipError_t err;
    err = hipGetLastError();
    if (err != hipSuccess)
	{
        printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}
    chronometer.stop();
    if(time_collision)
        *time_collision=chronometer.get_time();
    chronometer.play();
    hipMemcpy( gpu_ram_results, output_buffer, max_outputs*sizeof(char), hipMemcpyDeviceToHost);
    unsigned int index_a=total_objects-1;
    unsigned int index_b=0;
//    std::cout<<*((int*)(&(gpu_ram_results[0])))<<std::endl;
    for(unsigned int i=0;i<max_outputs;++i)
    {
        // std::cout<<"DATA "<<i<<": "<<int(gpu_ram_results[i])<<std::endl;
        if(gpu_ram_results[i])
        {
            total_collision[index_a]+=1;
            total_collision[index_b]+=1;
            collision.push_back(std::pair<int,int>(index_a,index_b));
        }
        ++index_b;
        if(index_a==index_b)
        {
            --index_a;
            index_b=0;
        }
    }
    // std::cout<<std::endl;
    //Collision END
    chronometer.stop();
    if(time_construction)
        *time_construction+=chronometer.get_time();
	return true;
}

int main(int argc,char* argv[])
{
	// std::cout<<"Interval: "<<sizeof(CInterval)<<std::endl;
	// std::cout<<"ListNode: "<<sizeof(ListNode)<<std::endl;
	// std::cout<<"List: "<<sizeof(List)<<std::endl;
	// std::cout<<"QueueNode: "<<sizeof(QueueNode)<<std::endl;
	// std::cout<<"Queue: "<<sizeof(Queue)<<std::endl;
	// std::cout<<"StackNode: "<<sizeof(StackNode)<<std::endl;
	// std::cout<<"Stack: "<<sizeof(Stack)<<std::endl;
	// std::cout<<"IntervalTree: "<<sizeof(IntervalTree)<<std::endl;
	// std::cout<<"RAMInterval: "<<sizeof(Interval)<<std::endl;
	// std::cout<<"RAMListNode: "<<sizeof(RAMListNode)<<std::endl;
	// std::cout<<"RAMList: "<<sizeof(RAMList)<<std::endl;
	// std::cout<<"RAMQueueNode: "<<sizeof(RAMQueueNode)<<std::endl;
	// std::cout<<"RAMQueue: "<<sizeof(RAMQueue)<<std::endl;
	// std::cout<<"RAMStackNode: "<<sizeof(RAMStackNode)<<std::endl;
	// std::cout<<"RAMStack: "<<sizeof(RAMStack)<<std::endl;
	// std::cout<<"RAMIntervalTreeNode: "<<sizeof(RAMIntervalTreeNode)<<std::endl;
	// std::cout<<"RAMIntervalTree: "<<sizeof(RAMIntervalTree)<<std::endl;
    bool print_collision=false;
    bool collision=false;
    bool render_frames=false;
    bool print_frames=true;
    LL::random_generate_new_seed();
    bool create_new_map=false;
    int mision=0;
    int max_test=-1;
    bool command=false;
    bool command_2=false;
    bool unisize=false;
    int total=-1;
    int threads=1;
    int parallel_x=1;
    int parallel_y=1;
    bool autosave=true;
	size_cuda_heap=1024*1024*size_cuda_heap;
	hipDeviceSetLimit(hipLimitMallocHeapSize, size_cuda_heap);
	hipDeviceGetLimit(&size_cuda_heap, hipLimitMallocHeapSize);
	std::cout<<"Size Heap: "<<size_cuda_heap<<std::endl;
    if(argc >= 6)
    {
        create_new_map=LL::to_int(argv[1]);
        mision=LL::to_int(argv[2]);
        max_test=LL::to_int(argv[3]);
        collision=LL::to_int(argv[4]);
        autosave=LL::to_int(argv[5]);
        int counter=6;
        if(create_new_map)
        {
            if(argc>counter)
            {
                unisize=LL::to_int(argv[counter++]);
                total=LL::to_int(argv[counter++]);
                command=true;
            }
        }
        if(argc>counter)
        {
            parallel_x=LL::to_int(argv[counter++]);
            parallel_y=LL::to_int(argv[counter++]);
            threads=LL::to_int(argv[counter++]);
            command_2=true;
        }
    }
    else if(argc==1)
    {
        std::cout<<"Nuevo Mapa (1/0): ";
        std::cin>>create_new_map;
        std::cout<<"AutoSave (1/0): ";
        std::cin>>autosave;
    }
    else
    {
        std::cout<<"SaP <CREATE_NEW_MAP> <ALGORITHM> <TEST> <START> <AUTOSAVE> [<UNISIZE> <TOTAL>] ";
        std::cout<<"[<PARALLEL_X> <PARALLEL_Y> <THREADS>]"<<std::endl;
        return -1;
    }
    Scene scene("scene.txt");
    scene.load();
    if(create_new_map)
    {
        if(!command)
        {
            std::cout<<"Unisize (1/0): ";
            std::cin>>unisize;
        }
        while(total <= 0)
        {
            std::cout<<"Total de Elementos: ";
            std::cin>>total;
        }
        scene.create(unisize,total);
        scene.save();
    }
    if(max_test == 0)
        return 0;
    std::string name_function;
    bool (*collision_function_3)(
                               float*,
                               int*,
                               unsigned int,
                               std::vector<int>&,
                               std::list<std::pair<int,int>>&,
                               float*,
                               float*,
                               int,
                               unsigned int,
                               unsigned int,
                               unsigned int,
                               unsigned int
                               )=nullptr;
    while(1)
    {
        if(mision==11)
        {
            if(!command_2)
            {
                std::cout<<"Parallel X: ";
                std::cin>>parallel_x;
                std::cout<<"Parallel Y: ";
                std::cin>>parallel_y;
                std::cout<<"Threads: ";
                std::cin>>threads;
            }
            name_function="SAP GPU Parallel";
            collision_function_3=SAP_GPU_Parallel;
            scene.build_gpu(parallel_x,parallel_y);
            build_sap_gpu_parallel(scene.get_local_size(),scene.size(),scene.get_size_gpu_grid(),scene.size_x(),scene.size_y());
            break;
        }
        else if(mision==12)
        {
            if(!command_2)
            {
                std::cout<<"Parallel X: ";
                std::cin>>parallel_x;
                std::cout<<"Parallel Y: ";
                std::cin>>parallel_y;
                std::cout<<"Threads: ";
                std::cin>>threads;
            }
            name_function="SAP GPU LL ITREE Parallel";
            collision_function_3=SAP_GPU_LL_ITree_Parallel;
            scene.build_gpu(parallel_x,parallel_y);
            build_sap_gpu_parallel(scene.get_local_size(),scene.size(),scene.get_size_gpu_grid(),scene.size_x(),scene.size_y());
            break;
        }
        // else if(mision==13)
        // {
            // if(!command_2)
            // {
                // std::cout<<"Parallel X: ";
                // std::cin>>parallel_x;
                // std::cout<<"Parallel Y: ";
                // std::cin>>parallel_y;
                // std::cout<<"Threads: ";
                // std::cin>>threads;
            // }
            // name_function="SAP GPU LL ITREE Parallel using struct RAM";
            // collision_function_3=SAP_GPU__RAM_LL_ITree_Parallel;
            // scene.build_gpu(parallel_x,parallel_y);
            // build_sap_ram_gpu_parallel(scene.get_local_size(),scene.size(),scene.get_size_gpu_grid(),scene.size_x(),scene.size_y(),threads);
            // break;
        // }
        std::cout<<"Ingresar Funcion:\n";
        std::cout<<"11: SAP GPU Parallel\n";
		std::cout<<"12: SAP LL_IntervalTree GPU Parallel\n";
		// std::cout<<"13: SAP LL_IntervalTree RAM GPU Parallel\n";
        std::cout<<"Opcion:";
        std::cin>>mision;
    }
    std::vector<int> on_collision(scene.size(),false);
    LL_AL5::init_allegro();
    LL_AL5::primitives_addon();
    LL_AL5::text_addon();
    LL_AL5::Display display(LL_AL5::desktop_size_x,LL_AL5::desktop_size_y,SCENE_SIZE_X,SCENE_SIZE_Y);
    display.set_display_mode(ALLEGRO_FULLSCREEN_WINDOW);
    display.create();
    LL_AL5::Font font;
    font.set_path("comic.ttf");
    font.set_size(SCENE_SIZE_X/100.0);
    font.load_ttf_font();
    LL_AL5::Font another_font;
    another_font.set_path("comic.ttf");
    another_font.set_size(12.0);
    another_font.load_ttf_font();
    scene.set_font(&font);
    LL_AL5::KeyControl key_control;
    key_control.add_key("Polygon",ALLEGRO_KEY_P);
    key_control.add_key("Frames",ALLEGRO_KEY_F);
    key_control.add_key("Render",ALLEGRO_KEY_S);
    key_control.add_key("Collision",ALLEGRO_KEY_C);
    key_control.add_key("Print Collision",ALLEGRO_KEY_SPACE);
    key_control.add_key("Controls",ALLEGRO_KEY_L);
//    std::cout<<"Controls:"<<std::endl;
//    std::cout<<"Show/Hide Polygon: P"<<std::endl;
//    std::cout<<"Show/Hide Text: F"<<std::endl;
//    std::cout<<"Render Object: S"<<std::endl;
//    std::cout<<"Show Controls: L"<<std::endl;
//    std::cout<<"Make Collision: C"<<std::endl;
//    std::cout<<"Print Collision Info: SPACE"<<std::endl;
//    std::cout<<"--------------------"<<std::endl;
    int total_frames=0;
    LL::Chronometer time;
    std::list<std::pair<int,int>> collision_list;
    time.play();
    LL_AL5::Input input;
    input.register_display(display);
    input.keyboard_on();
    input.set_key_control(&key_control);
    LL_AL5::Color black;
    LL_AL5::Color green(0,255);
    LL_AL5::Color color;
    LL_AL5::Text fps_text;
    fps_text.set_font(&another_font);
    fps_text.set_color(black);
    fps_text.set_pos(10,10);
    fps_text="0 fps";
    LL_AL5::Text collision_text;
    collision_text.set_font(&another_font);
    collision_text.set_color(black);
    collision_text.set_pos(10,10+2*another_font.get_size());
    collision_text="0 s";
    LL_AL5::Text total_text;
    total_text.set_font(&another_font);
    total_text.set_color(black);
    total_text.set_pos(10,10+4*another_font.get_size());
    total_text=LL::to_string(scene.size());
    LL_AL5::Text max_time_text;
    max_time_text.set_font(&another_font);
    max_time_text.set_color(black);
    max_time_text.set_pos(10,10+9*another_font.get_size());
    max_time_text="0 s";
    LL_AL5::Text min_time_text;
    min_time_text.set_font(&another_font);
    min_time_text.set_color(black);
    min_time_text.set_pos(10,10+7*another_font.get_size());
    min_time_text="0 s";
    LL_AL5::Text prom_time_text;
    prom_time_text.set_font(&another_font);
    prom_time_text.set_color(black);
    prom_time_text.set_pos(10,10+11*another_font.get_size());
    prom_time_text="0 s";
    LL_AL5::Text total_test_text;
    total_test_text.set_font(&another_font);
    total_test_text.set_color(black);
    total_test_text.set_pos(10,10+14*another_font.get_size());
    total_test_text="0";
    double min_time=1000;
    double max_time=0;
    double acum=0;
    int test=0;
    std::list<std::pair<float,float>> tiempos;
    while(!input.get_display_status() && test!=max_test)
    {
        ++total_frames;
        display.clear();
        scene.draw_scene(&display);
        if(render_frames)
        {
            for(unsigned int i=0;i<scene.size();++i)
            {
                switch(on_collision[i])
                {
                    case 1:
                    {
                        color.red   = 0;
                        color.green = 0;
                        color.blue  = 255;
                        break;
                    }
                    case 0:
                    {
                        color.red   = 0;
                        color.green = 0;
                        color.blue  = 0;
                        break;
                    }
                    default:
                    {
                        color.red   =255;
                        color.green =0;
                        color.blue  =0;
                        break;
                    }
                }
                scene[i].draw_object(&display,green,color);
            }
            for(unsigned int i=0;i<scene.size();++i)
                scene[i].draw_text(&display,black);
        }
        if(print_frames)
        {
            display.draw(&fps_text,false);
            display.draw(&total_text,false);
            display.draw(&collision_text,false);
            display.draw(&min_time_text,false);
            display.draw(&max_time_text,false);
            display.draw(&prom_time_text,false);
            display.draw(&total_test_text,false);
        }
        display.refresh();
        if(input.get_event())
        {
            if(input["Collision"])
            {
                collision=!collision;
                input["Collision"]=false;
            }
            if(input["Print Collision"])
            {
                print_collision=!print_collision;
                input["Print Collision"]=false;
            }
            if(input["Polygon"])
            {
                draw_polygon=!draw_polygon;
                input["Polygon"]=false;
            }
            if(input["Controls"])
            {
                std::cout<<"Controls:"<<std::endl;
                std::cout<<"Show/Hide Polygon: P"<<std::endl;
                std::cout<<"Show/Hide Text: F"<<std::endl;
                std::cout<<"Render Object: S"<<std::endl;
                std::cout<<"Show Controls: L"<<std::endl;
                std::cout<<"Make Collision: C"<<std::endl;
                std::cout<<"Print Collision Info: SPACE"<<std::endl;
                std::cout<<"--------------------"<<std::endl;
                input["Controls"]=false;
            }
            if(input["Frames"])
            {
                print_frames=!print_frames;
                input["Frames"]=false;
            }
            if(input["Render"])
            {
                render_frames=!render_frames;
                input["Render"]=false;
            }
        }
        if(collision)
        {
            collision_list.clear();
            for(unsigned int i=0;i<scene.size();++i)
                on_collision[i]=0;
            float time_construction;
            float time_collision;
			bool data=false;
            if(collision_function_3)
            {
                data=(collision_function_3(scene.get_objects_gpu_grid(),
										   scene.get_sizes_gpu_grid(),
										   scene.size(),
										   on_collision,
										   collision_list,
										   &time_construction,
										   &time_collision,
										   threads,
										   scene.size_x(),
										   scene.size_y(),
										   scene.get_size_gpu_grid(),
										   scene.get_local_size()));
            }
			if(data)
			{
				float total_time=time_construction+time_collision;
				tiempos.push_back(std::pair<float,float>(time_construction,time_collision));
				total_test_text=LL::to_string(++test);
				collision_text=LL::to_string(total_time)+" s";
				acum+=total_time;
				if(min_time>total_time)
				{
					min_time=total_time;
					min_time_text=LL::to_string(min_time)+" s";
				}
				if(max_time<total_time)
				{
					max_time=total_time;
					max_time_text=LL::to_string(max_time)+" s";
				}
				prom_time_text=LL::to_string(acum/test)+" s";
			}
        }
        if(print_collision)
        {
            system("cls");
            std::cout<<"--------------------"<<std::endl;
            collision_list.sort();
            for(auto i=collision_list.begin();i!=collision_list.end();++i)
                std::cout<<"("<<(*i).first<<","<<(*i).second<<")"<<std::endl;
            std::cout<<"--------------------"<<std::endl;
            print_collision=false;
        }
        if(time.get_time()>1)
        {
            fps_text=LL::to_string(total_frames/time.get_time())+" fps";
            time.clear();
            total_frames=0;
        }
    }
    input.unregister_display();
    input.unregister_timer();
    if(test)
    {
        std::cout<<"_________________________________________________"<<std::endl;
        std::cout<<"Total:     "<<scene.size()<<std::endl;
        std::cout<<"Test:      "<<test<<std::endl;
        std::cout<<"Algorithm: "<<name_function<<std::endl;
        std::cout<<"_________________________________________________"<<std::endl;
        std::cout<<"Min:       "<<min_time<<std::endl;
        std::cout<<"Max:       "<<max_time<<std::endl;
        std::cout<<"Prom:      "<<acum/test<<std::endl;
        std::cout<<"_________________________________________________"<<std::endl;
        std::cout<<std::endl;
    }
    if(autosave)
    {
        LL::FileStream txt_times;
        std::string path_name=name_function+" (S="+LL::to_string(scene.size())+").csv";
        txt_times.set_path(path_name);
        txt_times.load();
        txt_times.clear_file();
        txt_times.insert_line(0,tiempos.size());
        unsigned int index=0;
        for(auto tiempo:tiempos)
        {
            txt_times[index]=LL::to_string(tiempo.first)+";"+LL::to_string(tiempo.second)+";";
            std::replace(txt_times[index].begin(),txt_times[index].end(),'.',',');
            ++index;
        }
        std::cout<<"Saving: "<<path_name<<std::endl;
        txt_times.save();
    }
	if(gpu_results)
	{
		delete(gpu_results);
		hipFree(input_buffer);
		hipFree(ref_buffer);
		hipFree(output_buffer);
	}
	if(gpu_ram_results)
	{
		delete(gpu_ram_results);
		hipFree(input_buffer);
		hipFree(ref_buffer);
		hipFree(output_buffer);
		hipFree(ram_buffer);
	}
    return 0;
}
